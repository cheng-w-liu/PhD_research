#include "hip/hip_runtime.h"
// FILE: ising3d.c

#include <string>
#include <stdio.h>
#include <math.h>
#include <stdlib.h> // Provides rand(), RAND_MAX
#include <assert.h>
#include <time.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using std::string;

#define BLOCKLx 4
#define BLOCKLy 4
#define BLOCKLz 4
#define	MyBit 1ULL
#define	N64bit 64

typedef unsigned long long int bit64;
typedef bit64 spin_t;

int L, D, N;
int init, istp, mstp, nbins;
double T;
double enrg, ma, m2, m4;

spin_t *spins;
int **nnbors;
float prob[13];

texture<float, 1, hipReadModeElementType> Boltz;

__global__ void init_rand(int L, unsigned long long int seed, hiprandState_t *states) {

	   int global_x = blockIdx.x * blockDim.x + threadIdx.x;
	   int global_y = blockIdx.y * blockDim.y + threadIdx.y;
	   int global_z = blockIdx.z * blockDim.z + threadIdx.z;
	   int global_n = global_z * L * L + global_y * L + global_x;

           hiprand_init(seed, global_n, global_n, &states[global_n]);	   
	   
	   __syncthreads();
} // init_rand


__device__ float ran(hiprandState* global_state, int global_n) {
	   hiprandState_t local_state = global_state[global_n];
	   float r = hiprand_uniform(&local_state);
	   global_state[global_n] = local_state;
	   return r;
} // ran



__global__ void display_dims() {

    if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
        if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.y == 0) {
            printf("gDim.x : %i \n", gridDim.x);
            printf("bDim.x : %i \n", blockDim.x);
        }
    } 
    __syncthreads();
} // display_dims



__global__ void mc_updates(int L, hiprandState* global_state, spin_t *d_spins, int offset) {
	   __shared__ spin_t local_spins[(BLOCKLx+2)*(BLOCKLy+2)*(BLOCKLz+2)];
	   int global_x, global_y, global_z, global_n;
	   int nn_global_x, nn_global_y, nn_global_z, nn_global_n;
	   int local_x, local_y, local_z, local_n;
	   spin_t sj, ss0, ss1, ss2, ss3, ss4, ss5, mask;
	   int xm, xp, ym, yp, zm, zp, L2 = L*L, b, dE;
	      
	   global_x = blockIdx.x * blockDim.x + threadIdx.x;
	   global_y = blockIdx.y * blockDim.y + threadIdx.y;
	   global_z = blockIdx.z * blockDim.z + threadIdx.z;
	   global_n = global_z * L2 + global_y * L + global_x;
	   
	   local_x = threadIdx.x + 1;
	   local_y = threadIdx.y + 1;
	   local_z = threadIdx.z + 1;
	   local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;

	   local_spins[local_n] = d_spins[global_n];

	   if (threadIdx.x == 0) {
	      nn_global_x = ((blockIdx.x-1+gridDim.x)%gridDim.x)*BLOCKLx + BLOCKLx-1;
	      nn_global_y = global_y;
	      nn_global_z = global_z;
	      nn_global_n = nn_global_z * L2 + nn_global_y * L + nn_global_x;
	      local_x = 0;
	      local_y = threadIdx.y + 1;
	      local_z = threadIdx.z + 1;
	      local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];
	   }

	   if (threadIdx.x == BLOCKLx-1) {
	      nn_global_x = ((blockIdx.x+1)%gridDim.x)*BLOCKLx;
	      nn_global_y = global_y;
	      nn_global_z = global_z;
	      nn_global_n = nn_global_z * L2 + nn_global_y * L + nn_global_x;
	      local_x = BLOCKLx+1;
	      local_y = threadIdx.y + 1;
	      local_z = threadIdx.z + 1;
	      local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];
	   }

	   if (threadIdx.y == 0) {
	      nn_global_x = global_x;
	      nn_global_y = ((blockIdx.y-1+gridDim.y)%gridDim.y)*BLOCKLy + BLOCKLy-1;
	      nn_global_z = global_z;
	      nn_global_n = nn_global_z * L2 + nn_global_y * L + nn_global_x;
	      local_x = threadIdx.x + 1;
	      local_y = 0;
	      local_z = threadIdx.z + 1;
	      local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];	   
	   }

	   if (threadIdx.y == BLOCKLy-1) {
	      nn_global_x = global_x;
	      nn_global_y = ((blockIdx.y+1)%gridDim.y)*BLOCKLy;
	      nn_global_z = global_z;
	      nn_global_n = nn_global_z * L2 + nn_global_y * L + nn_global_x;
	      local_x = threadIdx.x + 1;
	      local_y = BLOCKLy+1;
	      local_z = threadIdx.z + 1;
	      local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];	   
	   }	   	   

	   if (threadIdx.z == 0) {
	      nn_global_x = global_x;
	      nn_global_y = global_y;
	      nn_global_z = ((blockIdx.z-1+gridDim.z)%gridDim.z)*BLOCKLz + BLOCKLz-1;
	      nn_global_n = nn_global_z * L2 + nn_global_y * L + nn_global_x;
	      local_x = threadIdx.x + 1;
	      local_y = threadIdx.y + 1;
	      local_z = 0;
	      local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];	   
	   }	   	   

	   if (threadIdx.z == BLOCKLz-1) {
	      nn_global_x = global_x;
	      nn_global_y = global_y;
	      nn_global_z = ((blockIdx.z+1)%gridDim.z)*BLOCKLz;
	      nn_global_n = nn_global_z * L2 + nn_global_y * L + nn_global_x;
	      local_x = threadIdx.x + 1;
	      local_y = threadIdx.y + 1;
	      local_z = BLOCKLz+1;
	      local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];	   
	   }	   	   

	   __syncthreads();

	   local_x = threadIdx.x + 1;
	   local_y = threadIdx.y + 1;
	   local_z = threadIdx.z + 1;
	   local_n = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	   sj = local_spins[local_n];
	       
	   if ( (threadIdx.x + threadIdx.y + threadIdx.z + offset)%2 == 0 ) {

	       xm = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x-1;
	       xp = local_z * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x+1;
	       ym = local_z * (BLOCKLx+2)*(BLOCKLy+2) + (local_y-1) * (BLOCKLx+2) + local_x;
	       yp = local_z * (BLOCKLx+2)*(BLOCKLy+2) + (local_y+1) * (BLOCKLx+2) + local_x;
	       zm = (local_z-1) * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;
	       zp = (local_z+1) * (BLOCKLx+2)*(BLOCKLy+2) + local_y * (BLOCKLx+2) + local_x;

	       ss0 = sj ^ local_spins[xm];
	       ss1 = sj ^ local_spins[xp];
	       ss2 = sj ^ local_spins[ym];
	       ss3 = sj ^ local_spins[yp];
	       ss4 = sj ^ local_spins[zm];
	       ss5 = sj ^ local_spins[zp];

	       for (b = 0; b < N64bit; ++b) {
	           dE = 0;
		   // dE <--> 2 \sigma^B_i XOR \sigma^B_j - 1 
		   mask = (MyBit << b);
		   dE += (ss0 & mask) ? 1 : -1;
		   dE += (ss1 & mask) ? 1 : -1;
		   dE += (ss2 & mask) ? 1 : -1;
		   dE += (ss3 & mask) ? 1 : -1;
		   dE += (ss4 & mask) ? 1 : -1;
		   dE += (ss5 & mask) ? 1 : -1;
		   if ( ran(global_state, global_n) < tex1Dfetch(Boltz,dE+6) )
		       sj ^= mask;
	       } // b	       
               local_spins[local_n] = sj;

	       d_spins[global_n] = local_spins[local_n];
	       
	   } // end of "if (Idx.x + Idx.y + Idx.z + offset)%2 == 0"

	   __syncthreads();

} // mc_updates


// ========================================================================== //
void initialize();
void read_file();
void set_parameters();
void allocate_arrays();
void lattice();
void configuration();
void random_conf();
void read_conf();
void write_conf();
void probability();
void deallocate_arrays();
void clean();
void measure();
void write_data();
// ========================================================================== //


int main(int argc, char* argv[]) {
    hiprandState_t *devStates;
    spin_t *dev_spins;
    float *dev_prob;

    srand(time(NULL)); 
    initialize();

    dim3 block(BLOCKLx, BLOCKLy, BLOCKLz);
    dim3 grid(L/BLOCKLx, L/BLOCKLy, L/BLOCKLz);

    //display_dims<<<grid, block>>>();

    hipMalloc((void **)&dev_spins, N*sizeof(spin_t));
    hipMalloc((void **)&devStates, N*sizeof(hiprandState_t));     

    init_rand<<<grid, block>>>(L, rand(), devStates);

    hipMalloc((void **) &dev_prob, (4*D+1) * sizeof(float));
    hipMemcpy(dev_prob, prob, (4*D+1)*sizeof(float), hipMemcpyHostToDevice);
    hipBindTexture(NULL, Boltz, dev_prob, (4*D+1)*sizeof(float));
        
    hipMemcpy(dev_spins, spins, N*sizeof(spin_t), hipMemcpyHostToDevice);
    
    for (int i = 0; i < istp; ++i) {
    	mc_updates<<<grid, block>>>(L, devStates, dev_spins, 0);
	mc_updates<<<grid, block>>>(L, devStates, dev_spins, 1);
    }
    hipMemcpy(spins, dev_spins, N*sizeof(spin_t), hipMemcpyDeviceToHost);
    write_conf();

    for (int k = 0; k < nbins; ++k) {
        clean();
    	for (int i = 0; i < mstp; ++i) {
    	    mc_updates<<<grid, block>>>(L, devStates, dev_spins, 0);
	    mc_updates<<<grid, block>>>(L, devStates, dev_spins, 1);
            hipMemcpy(spins, dev_spins, N*sizeof(spin_t), hipMemcpyDeviceToHost);
	    measure();
	} // i-mstp
        write_data();
        write_conf();
    } // k-bin
    
    if (dev_prob != NULL) { hipFree(dev_prob); dev_prob = NULL; }
    if (devStates != NULL) { hipFree(devStates); devStates = NULL; }
    if (dev_spins != NULL) { hipFree(dev_spins); dev_spins = NULL; }

    deallocate_arrays();
   
    return 0;
} // main



void write_data()
{
    FILE *ofptr;
    double dmstp = (double) mstp;

    enrg /= dmstp;
    ma /= dmstp;
    m2 /= dmstp;
    m4 /= dmstp;

    ofptr = fopen("data.dat","a");
    fprintf(ofptr,"%12.8f  %12.8f  %12.8f  %12.8f \n", enrg, ma, m2, m4);
    fclose(ofptr);

} // write_data



void clean()
{
    enrg = ma = m2 = m4 = 0.0e0;
} // clean


void measure() {
    int E = 0, j, b;
    int m[N64bit];
    bit64 mask, ss1, ss3, ss5;
    double dN = (double) N, dm , local_ma, local_m2, local_m4, d64 = (double) N64bit;
     
    for (b = 0; b < N64bit; ++b) m[b] = 0;

    for (j = 0; j < N; ++j) {
        ss1 = spins[j] ^ spins[ nnbors[j][1] ];
	ss3 = spins[j] ^ spins[ nnbors[j][3] ];
	ss5 = spins[j] ^ spins[ nnbors[j][5] ];
	for (b = 0; b < N64bit; ++b) {
            mask = (MyBit << b);
	    m[b] += ( (spins[j] & mask) ? 1 : -1 ); 
	    E += ( (ss1 & mask) ? -1 : 1 );
	    E += ( (ss3 & mask) ? -1 : 1 );
	    E += ( (ss5 & mask) ? -1 : 1 );
        } // b
    } // j

    enrg += (-(double) E)/(dN * (double) N64bit);
        
    local_ma = local_m2 = local_m4 = 0.0;
    for (b = 0; b < N64bit; ++b) {
        dm = (double) m[b]/dN;
        local_ma += fabs(dm);
        local_m2 += pow(dm,2.0);
        local_m4 += pow(dm,4.0);
    } //b                                                                                                         
    ma += local_ma/d64;
    m2 += local_m2/d64;
    m4 += local_m4/d64;

} // measure



void initialize()
{
    // 1) read-in input parameters
    read_file();

    // 2) set simulation parameters
    set_parameters();

    // 3) allocate arrays
    allocate_arrays();

    // 4) generate the 2D lattice
    lattice();   

    // 5) generate the initial configuration
    configuration();

    // 6) construct the probbility table
    probability();
    
} // initialize



void probability()
{
    float beta = 1.0e0/T;

    // e <--> 2 \sigma^B_i XOR \sigma^B_j - 1 
    for (int e = -6; e <= 6; ++e)
       	prob[e+6] = exp(2.0 * beta * (float) e);

} // probability



void configuration()
{
    if (init == 0)
        random_conf();
    else
        read_conf();

} // configuration



void random_conf()
{
    for (int i = 0; i < N; ++i) {
        spins[i] = 0;
        for(int b = 0; b < N64bit; ++b) {
            if (((double)rand())/((double)RAND_MAX) > 0.5)
                spins[i] ^= (MyBit << b);
        } // b 
    } // i
    
} // random_conf



void read_conf()
{
    FILE *fptr;
    fptr = fopen("spins.dat", "rt");
    if (fptr == NULL) { printf("can not open spins.dat"); exit(0); }
    for (int i = 0; i < N; ++i)
        fscanf(fptr, "%llu", &spins[i]);
    fclose(fptr);

} // read_conf



void write_conf()
{
  FILE *ofptr;
  int i;
  ofptr = fopen("spins.dat","w");
  for (i = 0; i < N; ++i)
    fprintf(ofptr,"%llu\n",spins[i]);
  fclose(ofptr);

} // write_conf



void lattice()
{
    int L2 = L * L;

    for (int z0 = 0; z0 < L; ++z0) {
        for (int y0 = 0; y0 < L; ++y0) {
            for (int x0 = 0; x0 < L; ++x0) {
                int x1 = (x0-1+L)%L;
	        int x2 = (x0+1)%L;
	        int y1 = (y0-1+L)%L;
	        int y2 = (y0+1)%L;
		int z1 = (z0-1+L)%L;
		int z2 = (z0+1)%L;

	        int j = z0 * L2 + y0 * L + x0;

	        nnbors[j][0] = z0 * L2 + y0 * L + x1;
	        nnbors[j][1] = z0 * L2 + y0 * L + x2;
	        nnbors[j][2] = z0 * L2 + y1 * L + x0;
	        nnbors[j][3] = z0 * L2 + y2 * L + x0;
	        nnbors[j][4] = z1 * L2 + y0 * L + x0;
	        nnbors[j][5] = z2 * L2 + y0 * L + x0;

            } // x0
        } // y0
    } // z0

} // lattice



void deallocate_arrays()
{
    if (spins != NULL) { free(spins); spins = NULL; }

    for (int i = 0; i < N; ++i) 
        if (nnbors[i] != NULL) 
            free(nnbors[i]); 
    if (nnbors != NULL) 
        free(nnbors);
    nnbors = NULL;

} // deallocate_arrays



void allocate_arrays()
{
    spins = (spin_t *) malloc(N * sizeof(spin_t));

    nnbors = (int **) malloc(N*sizeof(int*));
    for (int i = 0; i < N; ++i)
        nnbors[i] = (int *) malloc(2*D*sizeof(int));
    
} // allocate_arrays



void set_parameters()
{
    N = (int) pow(L, D);

} // set_parameters



void read_file()
{
    FILE *fptr;
    fptr = fopen("input.in", "rt");
    if (fptr == NULL) { 
        printf("can not open input.in"); 
	exit(0); 
    }
    fscanf(fptr,"%i %i %lf", &D, &L, &T);
    fscanf(fptr,"%i %i %i %i", &init, &istp, &mstp, &nbins);
    fclose(fptr);

} // read_file
