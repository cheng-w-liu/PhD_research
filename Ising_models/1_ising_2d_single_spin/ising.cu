#include "hip/hip_runtime.h"
// FILE: ising2d.c

#include <string>
#include <stdio.h>
#include <math.h>
#include <stdlib.h> // Provides rand(), RAND_MAX
#include <assert.h>
#include <time.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using std::string;

#define BLOCKLx 8
#define BLOCKLy 8

int L, D, N;
int init, istp, mstp, nbins;
double T;
double enrg, ma, m2, m4;

int *spins;
int **nnbors;
float prob[9];

texture<float, 1, hipReadModeElementType> Boltz;

__global__ void init_rand(int L, unsigned long long int seed, hiprandState_t *states) {

	   int global_x = blockIdx.x * blockDim.x + threadIdx.x;
	   int global_y = blockIdx.y * blockDim.y + threadIdx.y;
	   int global_n = global_y * L + global_x;

           hiprand_init(seed, global_n, global_n, &states[global_n]);	   
	   
	   __syncthreads();
} // init_rand


__device__ float ran(hiprandState* global_state, int global_n) {
	   hiprandState_t local_state = global_state[global_n];
	   float r = hiprand_uniform(&local_state);
	   global_state[global_n] = local_state;
	   return r;
} // ran


__global__ void mc_updates(int L, hiprandState* global_state, int *d_spins, int offset) {
	   __shared__ int local_spins[(BLOCKLx+2)*(BLOCKLy+2)];
	   int global_x, global_y, global_n;
	   int nn_global_x, nn_global_y, nn_global_n;
	   int local_x, local_y, local_n;
	   int e;

	   //assert(gridDim.x * blockDim.x == L);
	      
	   global_x = blockIdx.x * blockDim.x + threadIdx.x;
	   global_y = blockIdx.y * blockDim.y + threadIdx.y;
	   global_n = global_y * L + global_x;
	   
	   local_x = threadIdx.x + 1;
	   local_y = threadIdx.y + 1;
	   local_n = local_y * (BLOCKLx+2) + local_x;
	   local_spins[local_n] = d_spins[global_n];

	   if (threadIdx.x == 0) {
	      nn_global_x = ((blockIdx.x-1+gridDim.x)%gridDim.x)*BLOCKLx + BLOCKLx-1;
	      nn_global_y = global_y;
	      nn_global_n = nn_global_y * L + nn_global_x;
	      local_x = 0;
	      local_y = threadIdx.y + 1;
	      local_n = local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];
	   }

	   if (threadIdx.x == BLOCKLx-1) {
	      nn_global_x = ((blockIdx.x+1)%gridDim.x)*BLOCKLx;
	      nn_global_y = global_y;
	      nn_global_n = nn_global_y * L + nn_global_x;
	      local_x = BLOCKLx+1;
	      local_y = threadIdx.y + 1;
	      local_n = local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];
	   }

	   if (threadIdx.y == 0) {
	      nn_global_x = global_x;
	      nn_global_y = ((blockIdx.y-1+gridDim.y)%gridDim.y)*BLOCKLy + BLOCKLy-1;
	      nn_global_n = nn_global_y * L + nn_global_x;
	      local_x = threadIdx.x + 1;
	      local_y = 0;
	      local_n = local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];	   
	   }

	   if (threadIdx.y == BLOCKLy-1) {
	      nn_global_x = global_x;
	      nn_global_y = ((blockIdx.y+1)%gridDim.y)*BLOCKLy;
	      nn_global_n = nn_global_y * L + nn_global_x;
	      local_x = threadIdx.x + 1;
	      local_y = BLOCKLy+1;
	      local_n = local_y * (BLOCKLx+2) + local_x;
	      local_spins[local_n] = d_spins[nn_global_n];	   
	   }	   	   

	   __syncthreads();

	   local_x = threadIdx.x + 1;
	   local_y = threadIdx.y + 1;
	   local_n = local_y * (BLOCKLx+2) + local_x;
	       
	   if ( (threadIdx.x + threadIdx.y + offset)%2 == 0 ) {

	       int lf = local_y * (BLOCKLx+2) + local_x - 1;
	       int rt = local_y * (BLOCKLx+2) + local_x + 1;
	       int up = (local_y - 1) * (BLOCKLx+2) + local_x;
	       int dw = (local_y + 1) * (BLOCKLx+2) + local_x;
	       
	       e=local_spins[local_n]*(local_spins[lf]+local_spins[rt]+local_spins[up]+local_spins[dw]);

	       if ( ran(global_state, global_n) < tex1Dfetch(Boltz,e+4) ) 
	           local_spins[local_n] *= -1;

   	       d_spins[global_n] = local_spins[local_n];	

	   } // end of "if (Idx.x + Idx.y + offset)%2 == 0"

	   __syncthreads();

} // mc_updates


// ========================================================================== //
void initialize();
void read_file();
void set_parameters();
void allocate_arrays();
void lattice();
void configuration();
void random_conf();
void read_conf();
void write_conf();
void probability();
void deallocate_arrays();
void clean();
void measure();
void write_data();
// ========================================================================== //


int main(int argc, char* argv[]) {
    hiprandState_t *devStates;
    int *dev_spins;
    float *dev_prob;

    srand(time(NULL)); 
    initialize();

    dim3 block(BLOCKLx, BLOCKLy);
    dim3 grid(L/BLOCKLx, L/BLOCKLy);    

    hipMalloc((void **)&dev_spins, N*sizeof(int));
    hipMalloc((void **)&devStates, N*sizeof(hiprandState_t));     

    init_rand<<<grid, block>>>(L, 762198, devStates);

    hipMalloc((void **) &dev_prob, (4*D+1) * sizeof(float));
    hipMemcpy(dev_prob, prob, (4*D+1)*sizeof(float), hipMemcpyHostToDevice);
    hipBindTexture(NULL, Boltz, dev_prob, (4*D+1)*sizeof(float));
        
    hipMemcpy(dev_spins, spins, N*sizeof(int), hipMemcpyHostToDevice);
    
    for (int i = 0; i < istp; ++i) {
    	mc_updates<<<grid, block>>>(L, devStates, dev_spins, 0);
	mc_updates<<<grid, block>>>(L, devStates, dev_spins, 1);
    }
    hipMemcpy(spins, dev_spins, N*sizeof(int), hipMemcpyDeviceToHost);
    write_conf();

    for (int k = 0; k < nbins; ++k) {
        clean();
    	for (int i = 0; i < mstp; ++i) {
    	    mc_updates<<<grid, block>>>(L, devStates, dev_spins, 0);
	    mc_updates<<<grid, block>>>(L, devStates, dev_spins, 1);
            hipMemcpy(spins, dev_spins, N*sizeof(int), hipMemcpyDeviceToHost);
	    measure();
	} // i-mstp
        write_data();
        write_conf();
    } // k-bin
    
    if (dev_prob != NULL) { hipFree(dev_prob); dev_prob = NULL; }
    if (devStates != NULL) { hipFree(devStates); devStates = NULL; }
    if (dev_spins != NULL) { hipFree(dev_spins); dev_spins = NULL; }

    deallocate_arrays();
   
    return 0;
} // main



void write_data()
{
    FILE *ofptr;
    double dmstp = (double) mstp;

    enrg /= dmstp;
    ma /= dmstp;
    m2 /= dmstp;
    m4 /= dmstp;

    ofptr = fopen("data.dat","a");
    fprintf(ofptr,"%12.8f  %12.8f  %12.8f  %12.8f \n", enrg, ma, m2, m4);
    fclose(ofptr);

} // write_data



void clean()
{
    enrg = ma = m2 = m4 = 0.0e0;
} // clean



void measure() {
    int E = 0, M = 0;
    double m;

    for (int j = 0; j < N; ++j) {
        E += spins[j] * ( spins[ nnbors[j][0] ] + spins[ nnbors[j][3] ] );
        M += spins[j];
    } // j

    m = ((double) M)/((double) N);

    enrg += (-(double) E)/((double) N);
    ma += fabs(m);
    m2 += pow(m,2);
    m4 += pow(m,4);

} // measure



void initialize()
{
    // 1) read-in input parameters
    read_file();

    // 2) set simulation parameters
    set_parameters();

    // 3) allocate arrays
    allocate_arrays();

    // 4) generate the 2D lattice
    lattice();   

    // 5) generate the initial configuration
    configuration();

    // 6) construct the probbility table
    probability();
    
} // initialize



void probability()
{
    float beta = 1.0e0/T;

    for (int e = -4; e <= 4; ++e)
       	prob[e+4] = exp(-2.0 * beta * (float) e);

} // probability



void configuration()
{
    if (init == 0)
        random_conf();
    else
        read_conf();

} // configuration



void random_conf()
{
    for (int i = 0; i < N; ++i) 
    	spins[i] = ((double) rand()/(double) RAND_MAX) > 0.5 ? 1 : -1;
    
} // random_conf



void read_conf()
{
    FILE *fptr;
    fptr = fopen("spins.dat", "rt");
    if (fptr == NULL) { printf("can not open spins.dat"); exit(0); }
    for (int i = 0; i < N; ++i)
        fscanf(fptr, "%d", &spins[i]);
    fclose(fptr);

} // read_conf



void write_conf()
{
  FILE *ofptr;
  int i;
  ofptr = fopen("spins.dat","w");
  for (i = 0; i < N; ++i)
    fprintf(ofptr,"%i\n",spins[i]);
  fclose(ofptr);

} // write_conf



void lattice()
{
    for (int y0 = 0; y0 < L; ++y0) {
        for (int x0 = 0; x0 < L; ++x0) {
            int x1 = (x0+1)%L;
	    int x2 = (x0-1+L)%L;
	    int y1 = (y0-1+L)%L;
	    int y2 = (y0+1)%L;
	    int j = y0 * L + x0;
	    nnbors[j][0] = y0 * L + x1;
	    nnbors[j][1] = y0 * L + x2;
	    nnbors[j][2] = y1 * L + x0;
	    nnbors[j][3] = y2 * L + x0;
        } // x0
    } // y0

} // lattice



void deallocate_arrays()
{
    if (spins != NULL) { free(spins); spins = NULL; }

    for (int i = 0; i < N; ++i) 
        if (nnbors[i] != NULL) 
            free(nnbors[i]); 
    if (nnbors != NULL) 
        free(nnbors);
    nnbors = NULL;

} // deallocate_arrays



void allocate_arrays()
{
    spins = (int *) malloc(N * sizeof(int));

    nnbors = (int **) malloc(N*sizeof(int*));
    for (int i = 0; i < N; ++i)
        nnbors[i] = (int *) malloc(N*sizeof(int));
    
} // allocate_arrays



void set_parameters()
{
    N = (int) pow(L, D);

} // set_parameters



void read_file()
{
    FILE *fptr;
    fptr = fopen("input.in", "rt");
    if (fptr == NULL) { 
        printf("can not open input.in"); 
	exit(0); 
    }
    fscanf(fptr,"%i %i %lf", &D, &L, &T);
    fscanf(fptr,"%i %i %i %i", &init, &istp, &mstp, &nbins);
    fclose(fptr);

} // read_file
